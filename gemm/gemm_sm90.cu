#include "hip/hip_runtime.h"
#include "gemm_sm90.h"

#include <cassert>
#include <math.h>
#include <stdint.h>
#include <stdio.h>

#include <hip/hip_runtime.h>

namespace {

using mbarrier_t = long long;

__device__ uint32_t smem_ptr_u32(const void* smem_ptr)
{
    return static_cast<uint32_t>(__cvta_generic_to_shared(smem_ptr));
}

template <uint32_t SMEM_M, uint32_t SMEM_N, uint32_t SMEM_K, uint32_t CTA_MODULUS>
struct TiledMultiplier
{
    uint32_t size_m, size_n, size_k;

    // TODO remove
    float const* a;
    float const* bT;
    float* c;

    const CUtensorMap* tensorMap_a;
    const CUtensorMap* tensorMap_bT;  // Transposed; column major
    const CUtensorMap* tensorMap_c;

    static constexpr uint32_t WG_M = 64;
    static constexpr uint32_t WG_N = 64;
    static constexpr uint32_t WG_K = 8;
    static constexpr uint32_t RING_BUFFER_SIZE = 1;
    static constexpr CUtensorMapSwizzle swizzle = CU_TENSOR_MAP_SWIZZLE_NONE;

    // One buffer of ring buffer.
    struct Buffers
    {
        float a_tile[SMEM_M * SMEM_K];
        float bT_tile[SMEM_N * SMEM_K];  // Transposed
    };

    struct Shared
    {
        Buffers buffers[RING_BUFFER_SIZE];
        float c_tile[SMEM_M * SMEM_N];
        mbarrier_t mbar[RING_BUFFER_SIZE];
    };

    __host__ __device__ static constexpr uint32_t smem_size()
    {
        return sizeof(Shared);
    }

    __host__ __device__ static constexpr uint32_t consumer_wg_count()
    {
        // If output matrix is cut into (WG_M, WG_N) blocks, one warpgroup handles one matrix block.
        static_assert(SMEM_M % WG_M == 0);
        static_assert(SMEM_N % WG_N == 0);
        return (SMEM_M / WG_M) * (SMEM_N / WG_N);
    }
    __host__ __device__ static constexpr uint32_t cta_size()
    {
        // 1 extra warpgroup for memory.
        return (1 + consumer_wg_count()) * 128;
    }

    // If output matrix is cut into (SMEM_M, SMEM_N) blocks, one CTA handles one matrix block.
    static __host__ __device__ uint32_t m_cta(uint32_t size_m)
    {
        assert(size_m % SMEM_M == 0);
        return size_m / SMEM_M;
    }

    static __host__ __device__ uint32_t n_cta(uint32_t size_n)
    {
        assert(size_n % SMEM_N == 0);
        return size_n / SMEM_N;
    }

    // Per-warpgroup accumulator, holding one (WG_M, WG_N) tile.
    struct WG_Accum
    {
        // TODO use wgmma
        static constexpr uint32_t regcount = WG_M * WG_N / 128u;
        float regs[regcount];
    };

    // Warpgroup-convergent code.
    // Accumulate data from shared memory. Multiply the block matrices
    //   a_tile[wg_m_offset : wg_m_offset + WG_M, wg_k_offset : wg_k_offset + WG_K]
    //   bT_tile[wg_n_offset : wg_n_offset + WG_N, wg_k_offset : wg_k_offset + WG_K]
    // and add to the (WG_M, WG_N) tile held in WG_Accum.
    __device__ void wg_accum_tile(WG_Accum& accum, const Buffers& buffers, uint32_t wg_m_offset, uint32_t wg_n_offset,
                                  uint32_t wg_k_offset, bool zero_output) const
    {
        const uint32_t lane = threadIdx.x % 128u;

        for (uint32_t local_k = 0; local_k < WG_K; ++local_k) {
            for (uint32_t r = 0; r < accum.regcount; ++r) {
                const uint32_t r_in_wg = r + accum.regcount * lane;
                const uint32_t local_m = r_in_wg / WG_N;
                const uint32_t local_n = r_in_wg % WG_N;
                const uint32_t outer_m = wg_m_offset + local_m;
                const uint32_t outer_n = wg_n_offset + local_n;
                const uint32_t outer_k = wg_k_offset + local_k;
                const float a_val = buffers.a_tile[outer_m * SMEM_K + outer_k];
                const float b_val = buffers.bT_tile[outer_n * SMEM_K + outer_k];
                if (zero_output && local_k == 0) {
                    accum.regs[r] = a_val * b_val;
                }
                else {
                    accum.regs[r] = fma(a_val, b_val, accum.regs[r]);
                }
            }
        }
    }

    // Warpgroup-convergent code
    // Write the (WG_M, WG_N) tile to shared.c_tile, at offset (wg_m_offset, wg_n_offset).
    __device__ void wg_accum_to_shared(Shared& shared, const WG_Accum& accum,
                                       uint32_t wg_m_offset, uint32_t wg_n_offset) const
    {
        const uint32_t lane = threadIdx.x % 128u;

        for (uint32_t r = 0; r < accum.regcount; ++r) {
            const uint32_t r_in_wg = r + accum.regcount * lane;
            const uint32_t local_m = r_in_wg / WG_N;
            const uint32_t local_n = r_in_wg % WG_N;
            const uint32_t outer_m = wg_m_offset + local_m;
            const uint32_t outer_n = wg_n_offset + local_n;
            shared.c_tile[outer_m * SMEM_N + outer_n] = accum.regs[r];
        }
    }

    // Fill shared memory A tile with SMEM_M×SMEM_K block starting at (cta_m_offset, cta_k_offset)
    // Fill shared memory B^T tile with SMEM_N×SMEM_K block starting at (cta_n_offset, cta_k_offset)
    __device__ void warp_async_load_block(Buffers& buffers, mbarrier_t& mbar, uint32_t cta_m_offset,
                                          uint32_t cta_n_offset, uint32_t cta_k_offset) const
    {
        const uint32_t lane = threadIdx.x % 32u;

        const bool use_tma = true;
        if (use_tma && lane == 0) {  // XXX lane == 0 is "wrong"
            asm volatile(
                "cp.async.bulk.tensor.2d.shared::cluster.global.tile.mbarrier::complete_tx::bytes"
                " [%0], [%1, {%3, %4}], [%2];"
                :
                : "r"(smem_ptr_u32(&buffers.a_tile)),
                  "l"(tensorMap_a),
                  "r"(smem_ptr_u32(&mbar)),
                  "r"(cta_k_offset), "r"(cta_m_offset)
                : "memory");
            asm volatile(
                "cp.async.bulk.tensor.2d.shared::cluster.global.tile.mbarrier::complete_tx::bytes"
                " [%0], [%1, {%3, %4}], [%2];"
                :
                : "r"(smem_ptr_u32(&buffers.bT_tile)),
                  "l"(tensorMap_bT),
                  "r"(smem_ptr_u32(&mbar)),
                  "r"(cta_k_offset), "r"(cta_n_offset)
                : "memory");
            const uint32_t expect_count = (SMEM_M + SMEM_N) * SMEM_K * sizeof(float);
            uint64_t mbar_state;
            asm volatile(
                "mbarrier.arrive.expect_tx.shared::cta.b64 %0, [%1], %2;"
                : "=l"(mbar_state)
                : "r"(smem_ptr_u32(&mbar)), "r"(expect_count));
            asm volatile(
                "{.reg.pred P1; BEFORE_WAIT: mbarrier.try_wait.acquire.cta.shared::cta.b64 P1, [%0], %1; @P1 bra.uni WAIT_DONE; bra.uni BEFORE_WAIT; WAIT_DONE: }"
                :
                : "r"(smem_ptr_u32(&mbar)), "l"(mbar_state));
        }
    }

    // Static assignment of warpgroups within CTA to per-warpgroup output tiles (WG_M, WG_N) within
    // per-CTA output tile (SMEM_M, SMEM_N), plus one extra warpgroup for memory transactions.
    __device__ bool is_memory_wg() const
    {
        return (threadIdx.x / 128u) == consumer_wg_count();
    }

    __device__ uint32_t get_wg_m_idx() const
    {
        const uint32_t wg_index = threadIdx.x / 128u;
        assert(wg_index < consumer_wg_count());
        return wg_index / (SMEM_N / WG_N);
    }

    __device__ uint32_t get_wg_n_idx() const
    {
        const uint32_t wg_index = threadIdx.x / 128u;
        return wg_index % (SMEM_N / WG_N);
    }

    __device__ void cta_first_time_init(Shared& shared) const
    {
        for (uint32_t i = threadIdx.x; i < RING_BUFFER_SIZE; i += blockDim.x) {
            asm volatile("mbarrier.init.shared::cta.b64 [%0], 1;" :: "r"(smem_ptr_u32(&shared.mbar[i])));
            asm volatile("fence.proxy.async;");
        }
    }

    // CTA cooperates to fill the output matrix block of size (SMEM_M, SMEM_N) starting at (cta_m_offset, cta_n_offset).
    // Requires smem-allocated ring buffer.
    __device__ void cta_compute_block(uint32_t cta_m_offset, uint32_t cta_n_offset, Shared& shared) const
    {
        assert(cta_m_offset % SMEM_M == 0);
        assert(cta_n_offset % SMEM_N == 0);
        assert(size_k % SMEM_K == 0);
        const uint32_t k_blk_dim = size_k / SMEM_K;

        WG_Accum accum;

        for (uint32_t cta_k_idx = 0; cta_k_idx < k_blk_dim; ++cta_k_idx) {
            if (is_memory_wg()) {
                if (threadIdx.x % 128u < 32u) {
                    const auto cta_k_offset = cta_k_idx * SMEM_K;
                    warp_async_load_block(shared.buffers[0], shared.mbar[0], cta_m_offset, cta_n_offset, cta_k_offset);
                }
            }
            __syncthreads();  // TODO
            if (!is_memory_wg()) {
                const uint32_t wg_m_offset = get_wg_m_idx() * WG_M;
                const uint32_t wg_n_offset = get_wg_n_idx() * WG_N;
                for (uint32_t wg_k_idx = 0; wg_k_idx < SMEM_K / WG_K; ++wg_k_idx) {
                    const uint32_t wg_k_offset = wg_k_idx * WG_K;
                    const bool zero_accum = cta_k_idx == 0 && wg_k_offset == 0;
                    wg_accum_tile(accum, shared.buffers[0], wg_m_offset, wg_n_offset, wg_k_offset, zero_accum);
                }
            }
            __syncthreads();  // TODO
        }

        if (!is_memory_wg()) {
            const uint32_t wg_m_offset = get_wg_m_idx() * WG_M;
            const uint32_t wg_n_offset = get_wg_n_idx() * WG_N;
            wg_accum_to_shared(shared, accum, wg_m_offset, wg_n_offset);
        }

        __syncthreads();  // TODO

        for (uint32_t m = 0; m < SMEM_M; m++) {
            for (uint32_t n = threadIdx.x; n < SMEM_N; n += blockDim.x) {
                const uint32_t global_m = m + cta_m_offset;
                const uint32_t global_n = n + cta_n_offset;
                c[global_m * size_n + global_n] = shared.c_tile[m * SMEM_N + n];
            }
        }
    }

    __device__ void kernel_main()
    {
        assert(gridDim.x == m_cta(size_m) * n_cta(size_n));
        assert(blockDim.x == cta_size());

        const uint32_t cta_rows = size_m / SMEM_M;
        const uint32_t cta_cols = size_n / SMEM_N;
        const uint32_t cta_col_remainder = cta_cols % CTA_MODULUS;
        const uint32_t superblock_count = cta_cols / CTA_MODULUS;
        const uint32_t superblock_cta_count = cta_rows * CTA_MODULUS;
        const uint32_t superblock_idx = blockIdx.x / superblock_cta_count;
        const uint32_t cta_index_in_superblock = blockIdx.x % superblock_cta_count;

        uint32_t cta_m_idx, cta_n_idx;

        if (superblock_idx < superblock_count) {
            cta_m_idx = cta_index_in_superblock / CTA_MODULUS;
            cta_n_idx = cta_index_in_superblock % CTA_MODULUS + CTA_MODULUS * superblock_idx;
        }
        else {
            assert(superblock_idx == superblock_count);
            cta_m_idx = cta_index_in_superblock / cta_col_remainder;
            cta_n_idx = cta_index_in_superblock % cta_col_remainder + CTA_MODULUS * superblock_idx;
        }
        assert(cta_m_idx < cta_rows);
        assert(cta_n_idx < cta_cols);

        extern __shared__ char smem[];
        cta_first_time_init(reinterpret_cast<Shared&>(*smem));
        __syncthreads();
        cta_compute_block(cta_m_idx * SMEM_M, cta_n_idx * SMEM_N, reinterpret_cast<Shared&>(*smem));
    }

    static void init_tensorMap(CUtensorMap* tensorMap, const float* globalAddress, uint32_t rows, uint32_t cols,
                               uint32_t smem_rows, uint32_t smem_cols)
    {
        const CUtensorMapDataType tensorDataType = CU_TENSOR_MAP_DATA_TYPE_FLOAT32;
        const uint32_t tensorRank = 2;
        const cuuint64_t globalDim[2] = {cols, rows};
        const cuuint64_t globalStrides[1] = {4*cols};
        const cuuint32_t boxDim[2] = {smem_cols, smem_rows};
        const cuuint32_t elementStrides[2] = {1, 1};
        const CUtensorMapInterleave interleave = CU_TENSOR_MAP_INTERLEAVE_NONE;
        const CUtensorMapL2promotion l2Promotion = CU_TENSOR_MAP_L2_PROMOTION_NONE;
        const CUtensorMapFloatOOBfill oobFill = CU_TENSOR_MAP_FLOAT_OOB_FILL_NAN_REQUEST_ZERO_FMA;

        const hipError_t result = cuTensorMapEncodeTiled(
                tensorMap,
                tensorDataType,
                tensorRank,
                const_cast<float*>(globalAddress),
                globalDim,
                globalStrides,
                boxDim,
                elementStrides,
                interleave,
                swizzle,
                l2Promotion,
                oobFill);
        if (result != 0) {
            fprintf(stderr, "cuTensorMapEncodeTiled: %i\n", (int)result);
            assert(0);
        }
    }

    static void launch(hipStream_t stream, uint32_t size_m, uint32_t size_n, uint32_t size_k,
                       const float* a, const float* b, float* c);
};

template <typename Multiplier>
__global__ void
__launch_bounds__(Multiplier::cta_size())
tiled_multiplier_kernel(uint32_t size_m, uint32_t size_n, uint32_t size_k,
                        const float* a, const float* bT, float* c,
                        __grid_constant__ const CUtensorMap tensorMap_a,
                        __grid_constant__ const CUtensorMap tensorMap_bT,
                        __grid_constant__ const CUtensorMap tensorMap_c)
{
    Multiplier multiplier{size_m, size_n, size_k, a, bT, c, &tensorMap_a, &tensorMap_bT, &tensorMap_c};
    multiplier.kernel_main();
}

template <uint32_t SMEM_M, uint32_t SMEM_N, uint32_t SMEM_K, uint32_t CTA_MODULUS>
void TiledMultiplier<SMEM_M, SMEM_N, SMEM_K, CTA_MODULUS>::launch(
        hipStream_t stream, uint32_t size_m, uint32_t size_n, uint32_t size_k,
        const float* a, const float* bT, float* c)
{
    using Multiplier = TiledMultiplier<SMEM_M, SMEM_N, SMEM_K, CTA_MODULUS>;

    CUtensorMap tensorMap_a, tensorMap_bT, tensorMap_c;
    init_tensorMap(&tensorMap_a, a, size_m, size_k, SMEM_M, SMEM_K);
    init_tensorMap(&tensorMap_bT, bT, size_n, size_k, SMEM_N, SMEM_K);
    init_tensorMap(&tensorMap_c, c, size_m, size_n, SMEM_M, SMEM_N);

    const uint32_t grid = m_cta(size_m) * n_cta(size_n);
    const uint32_t block = cta_size();
    const uint32_t smem = smem_size();
    hipFuncSetAttribute(reinterpret_cast<const void*>(tiled_multiplier_kernel<Multiplier>), hipFuncAttributeMaxDynamicSharedMemorySize, smem);
    tiled_multiplier_kernel<Multiplier> <<<grid, block, smem, stream>>>(size_m, size_n, size_k, a, bT, c,
                                                                        tensorMap_a, tensorMap_bT, tensorMap_c);
}

}  // end namespace

void matmul_sm90(GPU_Tensors t, hipStream_t stream)
{
    constexpr uint32_t smem_m = 128;
    constexpr uint32_t smem_n = 128;
    constexpr uint32_t smem_k = 16;
    constexpr uint32_t cta_modulus = 4;

    const uint32_t size_m = t.M;
    const uint32_t size_n = t.N;
    const uint32_t size_k = t.K;

    if (size_m % smem_m == 0 && size_n % smem_n == 0 && size_k % smem_k == 0) {
        TiledMultiplier<smem_m, smem_n, smem_k, cta_modulus>::launch(stream, size_m, size_n, size_k, t.a, t.b, t.c);
    }
    else {
        assert(0);
    }
}
