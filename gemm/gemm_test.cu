#include "hip/hip_runtime.h"
#include "gemm_test.h"

#include <algorithm>
#include <stdlib.h>
#include <stdio.h>

#include "gemm_sm80.h"
#include "gemm_sm90.h"

namespace {

// Copied pseudo random number generation code.
// http://www.jcgt.org/published/0009/03/02/
// Hash Functions for GPU Rendering, Mark Jarzynski, Marc Olano, NVIDIA
__device__ uint64_t pcg3d(uint32_t x, uint32_t y, uint32_t z)
{
  x = x*1664525u + 1013904223u;
  y = y*1664525u + 1013904223u;
  z = z*1664525u + 1013904223u;

  x += y*z;
  y += z*x;
  z += x*y;

  x ^= x >> 16u;
  y ^= y >> 16u;
  z ^= z >> 16u;

  x += y*z;
  y += z*x;
  z += x*y;

  return x ^ uint64_t(y) << 12u ^ uint64_t(z) << 24u;
}

__global__ void device_init_test_data(float* d_tensor, uint32_t rows, uint32_t cols,
                                      TestDataCode code, bool transpose_rule)
{
    uint32_t tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    for (uint32_t r = tid_y; r < rows; r += blockDim.y * gridDim.y) {
        for (uint32_t c = tid_x; c < cols; c += blockDim.x * gridDim.x) {
            float value;
            const uint32_t x = transpose_rule ? r : c;
            const uint32_t y = transpose_rule ? c : r;
            switch (code) {
              case TestDataCode::identity:
                value = x == y ? 1.0f : 0.0f;
                break;
              case TestDataCode::tiled_numbers:
                value = (x % 64) + 100.0f * (y % 64);
                break;
              case TestDataCode::random:
              default:
                value = float(pcg3d(x, y, 19980724) % 20010106u);
                break;
            }
            d_tensor[r * cols + c] = value;
        }
    }
}

__global__ void device_compare_tensor_test_init_bitfield(unsigned long long* d_bitfield)
{
    *d_bitfield = UINT64_MAX;
}

// Requires that *d_bitfield is initialized to UINT64_MAX.
// Compare the two equal-sized matrices and, if any comparison failures, put the coordinates of the wrong value
// into *d_bitfield, packed as (row << 32 | col).
__global__ void device_compare_tensor_test(const float* d_a, const float* d_b, uint32_t rows, uint32_t cols,
                                           unsigned long long* d_bitfield)
{
    uint32_t tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    for (uint32_t y = tid_y; y < rows; y += blockDim.y * gridDim.y) {
        for (uint32_t x = tid_x; x < cols; x += blockDim.x * gridDim.x) {
            float a = d_a[y * cols + x];
            float b = d_b[y * cols + x];
            bool correct = a * b >= 0.0f;  // Sign error, or inf/nan if wrong
            if (correct) {
                a = fabsf(a);
                b = fabsf(b);
                const float m = fminf(a, b);
                const float M = fmaxf(a, b);
                correct = M == 0 || M / m < (1.0f + 1/32.0f);
            }
            if (!correct) {
                unsigned long long packed_coords = uint64_t(y) << 32u | x;
                atomicMin(d_bitfield, packed_coords);
            }
        }
    }
}

// Print info on wrong value from function above.
__global__ void device_compare_tensor_test_print(TestParams test_params,
                                                 const float* d_a, const float* d_b, uint32_t rows, uint32_t cols,
                                                 unsigned long long* d_bitfield)
{
    uint32_t y = uint32_t(*d_bitfield >> 32u);
    uint32_t x = uint32_t(*d_bitfield);

    if (x < cols && y < rows) {
        const float a = d_a[y * cols + x];
        const float b = d_b[y * cols + x];
        printf("TestParams{%u,%u,%u, %i,%i} [%u,%u] %g != %g\n", test_params.M, test_params.N, test_params.K,
               static_cast<int>(test_params.test_data_code_A), static_cast<int>(test_params.test_data_code_B),
               y, x, a, b);
        uint32_t y_min = y < 2 ? 0u : y - 2;
        uint32_t y_max = y + 2 >= rows ? rows - 1u : y + 2;
        uint32_t x_min = x < 2 ? 0u : x - 2;
        uint32_t x_max = x + 2 >= cols ? cols - 1u : x + 2;

        for (uint32_t cy = y_min; cy <= y_max; cy++) {
            for (uint32_t cx = x_min; cx <= x_max; cx++) {
                if (cy == y && cx == x) {
                    printf("\x1b[1m");
                }
                printf("[%6g, %5g]  ", d_a[cy*cols + cx], d_b[cy*cols + cx]);
                if (cy == y && cx == x) {
                    printf("\x1b[0m");
                }
            }
            printf("\n");
        }
    }
}

void launch_device_compare_tensor(TestParams test_params,
                                  const float* d_a, const float* d_b, uint32_t rows, uint32_t cols,
                                  unsigned long long* d_bitfield, hipStream_t stream)
{
    dim3 grid{(cols + 15u) / 16u, (rows + 15u) / 16u, 1};
    dim3 block{16, 16, 1};
    device_compare_tensor_test_init_bitfield<<<1, 1, 0, stream>>>(d_bitfield);
    device_compare_tensor_test<<<grid, block, 0, stream>>>(d_a, d_b, rows, cols, d_bitfield);
    device_compare_tensor_test_print<<<1, 1, 0, stream>>>(test_params, d_a, d_b, rows, cols, d_bitfield);
}

}  // end namespace

void gemm_test(TestParams params, hipStream_t stream)
{
    unsigned long long* d_bitfield = nullptr;
    float* d_a = nullptr;
    float* d_b = nullptr;
    float* d_bT = nullptr;
    float* d_c_sm80 = nullptr;
    float* d_c_sm90_warmup = nullptr;
    float* d_c_sm90_tested = nullptr;

    hipMallocAsync(&d_bitfield, sizeof(unsigned long long), stream);
    hipMallocAsync(&d_a, params.M * params.K * sizeof(float), stream);
    hipMallocAsync(&d_b, params.N * params.K * sizeof(float), stream);
    hipMallocAsync(&d_bT, params.N * params.K * sizeof(float), stream);
    hipMallocAsync(&d_c_sm80, params.M * params.N * sizeof(float), stream);
    hipMallocAsync(&d_c_sm90_warmup, params.M * params.N * sizeof(float), stream);
    hipMallocAsync(&d_c_sm90_tested, params.M * params.N * sizeof(float), stream);

    if (!d_bitfield || !d_a || !d_b || !d_c_sm80 || !d_c_sm90_warmup || !d_c_sm90_tested) {
        fprintf(stderr, "Out of GPU memory\n");
        exit(1);
    }

    // Initialize "random" test data
    {
        dim3 grid_a{(params.K + 15u) / 16u, (params.M + 15u) / 16u, 1};
        dim3 grid_b{(params.N + 15u) / 16u, (params.K + 15u) / 16u, 1};
        dim3 block{16, 16, 1};
        device_init_test_data<<<grid_a, block, 0, stream>>>(d_a, params.M, params.K, params.test_data_code_A, false);
        device_init_test_data<<<grid_b, block, 0, stream>>>(d_b, params.K, params.N, params.test_data_code_B, false);
        device_init_test_data<<<grid_b, block, 0, stream>>>(d_bT, params.N, params.K, params.test_data_code_B, true);
    }

    auto fill_garbage = [params, stream] (float* d_c)
    {
        hipMemsetAsync(d_c, 0xDD, sizeof(*d_c) * params.M * params.N, stream);
    };

    // Initialize SM80 data
    {
        GPU_Tensors t{params.M, params.N, params.K, d_a, d_b, d_c_sm80, 0, 0, 0};
        fill_garbage(t.c);
        matmul_sm80(t, stream);
    }

    // Initialize SM90 data
    {
        GPU_Tensors t{params.M, params.N, params.K, d_a, d_bT, d_c_sm90_warmup, 0, 1, 0};
        fill_garbage(t.c);
        matmul_sm90(t, stream);
    }
    launch_device_compare_tensor(params, d_c_sm80, d_c_sm90_warmup, params.M, params.N, d_bitfield, stream);

    // Test loop
    constexpr uint32_t test_count = 15;
    float test_times[test_count] = {};
    hipEvent_t test_events[test_count + 1];
    auto new_event = [stream]
    {
        hipEvent_t event{};
        if (const hipError_t err = hipEventCreate(&event)) {
            fprintf(stderr, "hipError_t %i: %s\n", (int)err, hipGetErrorString(err));
            exit(1);
        }
        hipEventRecord(event, stream);
        return event;
    };
    for (uint32_t test_i = 0; test_i < test_count; ++test_i) {
        if (test_i == 0) {
            test_events[0] = new_event();
        }
        GPU_Tensors t{params.M, params.N, params.K, d_a, d_bT, d_c_sm90_tested, 0, 1, 0};
        matmul_sm90(t, stream);
        test_events[test_i + 1] = new_event();
    }
    hipStreamSynchronize(stream);
    for (uint32_t test_i = 0; test_i < test_count; ++test_i) {
        hipEventElapsedTime(&test_times[test_i], test_events[test_i], test_events[test_i + 1]);
        hipEventDestroy(test_events[test_i]);
    }
    hipEventDestroy(test_events[test_count]);
    std::sort(&test_times[0], &test_times[test_count]);
    printf("TestParams{%u,%u,%u, %i,%i} %.3g ms\n", params.M, params.N, params.K,
           static_cast<int>(params.test_data_code_A), static_cast<int>(params.test_data_code_B),
           test_times[test_count / 4]);

    launch_device_compare_tensor(params, d_c_sm80, d_c_sm90_tested, params.M, params.N, d_bitfield, stream);

    hipFreeAsync(d_a, stream);
    hipFreeAsync(d_b, stream);
    hipFreeAsync(d_bT, stream);
    hipFreeAsync(d_c_sm80, stream);
    hipFreeAsync(d_c_sm90_warmup, stream);
    hipFreeAsync(d_c_sm90_tested, stream);

    hipStreamSynchronize(stream);
    if (const hipError_t err = hipGetLastError()) {
        fprintf(stderr, "hipError_t %i: %s\n", (int)err, hipGetErrorString(err));
        exit(1);
    }
}
