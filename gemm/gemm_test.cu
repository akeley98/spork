#include "hip/hip_runtime.h"
#include "gemm_test.h"

#include <stdlib.h>
#include <stdio.h>

#include "gemm_sm80.h"
#include "exo_par.h"

namespace {

// Copied pseudo random number generation code.
// http://www.jcgt.org/published/0009/03/02/
// Hash Functions for GPU Rendering, Mark Jarzynski, Marc Olano, NVIDIA
__device__ uint64_t pcg3d(uint32_t x, uint32_t y, uint32_t z)
{
  x = x*1664525u + 1013904223u;
  y = y*1664525u + 1013904223u;
  z = z*1664525u + 1013904223u;

  x += y*z;
  y += z*x;
  z += x*y;

  x ^= x >> 16u;
  y ^= y >> 16u;
  z ^= z >> 16u;

  x += y*z;
  y += z*x;
  z += x*y;

  return x ^ uint64_t(y) << 12u ^ uint64_t(z) << 24u;
}

__global__ void device_init_test_data(float* d_tensor, uint32_t rows, uint32_t cols, TestDataCode code)
{
    uint32_t tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    for (uint32_t y = tid_y; y < rows; y += blockDim.y * gridDim.y) {
        for (uint32_t x = tid_x; x < cols; x += blockDim.x * gridDim.x) {
            float value;
            switch (code) {
              case TestDataCode::identity:
                value = x == y ? 1.0f : 0.0f;
                break;
              case TestDataCode::tiled_numbers:
                value = (x % 64) + 100.0f * (y % 64);
                break;
              case TestDataCode::random:
              default:
                value = float(pcg3d(x, y, 19980724) % 20010106u);
                break;
            }
            d_tensor[y * cols + x] = value;
        }
    }
}

__global__ void device_compare_tensor_test_init_bitfield(unsigned long long* d_bitfield)
{
    *d_bitfield = UINT64_MAX;
}

// Requires that *d_bitfield is initialized to UINT64_MAX.
// Compare the two equal-sized matrices and, if any comparison failures, put the coordinates of the wrong value
// into *d_bitfield, packed as (row << 32 | col).
__global__ void device_compare_tensor_test(const float* d_a, const float* d_b, uint32_t rows, uint32_t cols,
                                           unsigned long long* d_bitfield)
{
    uint32_t tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    for (uint32_t y = tid_y; y < rows; y += blockDim.y * gridDim.y) {
        for (uint32_t x = tid_x; x < cols; x += blockDim.x * gridDim.x) {
            float a = d_a[y * cols + x];
            float b = d_b[y * cols + x];
            bool correct = a * b >= 0.0f;  // Sign error, or inf/nan if wrong
            if (correct) {
                a = fabsf(a);
                b = fabsf(b);
                const float m = fminf(a, b);
                const float M = fmaxf(a, b);
                correct = M == 0 || M / m < (1.0f + 1/32.0f);
            }
            if (!correct) {
                unsigned long long packed_coords = uint64_t(y) << 32u | x;
                atomicMin(d_bitfield, packed_coords);
            }
        }
    }
}

// Print info on wrong value from function above.
__global__ void device_compare_tensor_test_print(TestParams test_params,
                                                 const float* d_a, const float* d_b, uint32_t rows, uint32_t cols,
                                                 unsigned long long* d_bitfield)
{
    uint32_t y = uint32_t(*d_bitfield >> 32u);
    uint32_t x = uint32_t(*d_bitfield);

    if (x < cols && y < rows) {
        const float a = d_a[y * cols + x];
        const float b = d_b[y * cols + x];
        printf("TestParams{%u,%u,%u, %i,%i} [%u,%u] %g != %g\n", test_params.M, test_params.N, test_params.K,
               static_cast<int>(test_params.test_data_code_A), static_cast<int>(test_params.test_data_code_B),
               y, x, a, b);
    }
}

void launch_device_compare_tensor(TestParams test_params,
                                  const float* d_a, const float* d_b, uint32_t rows, uint32_t cols,
                                  unsigned long long* d_bitfield, hipStream_t stream)
{
    dim3 grid{(cols + 15u) / 16u, (rows + 15u) / 16u, 1};
    dim3 block{16, 16, 1};
    device_compare_tensor_test_init_bitfield<<<1, 1, 0, stream>>>(d_bitfield);
    device_compare_tensor_test<<<grid, block, 0, stream>>>(d_a, d_b, rows, cols, d_bitfield);
    device_compare_tensor_test_print<<<1, 1, 0, stream>>>(test_params, d_a, d_b, rows, cols, d_bitfield);
}

}  // end namespace

void gemm_test(TestParams params, hipStream_t stream)
{
    unsigned long long* d_bitfield = nullptr;
    float* d_a = nullptr;
    float* d_b = nullptr;
    float* d_c_sm80 = nullptr;
    float* d_c_sm90_warmup = nullptr;
    float* d_c_sm90_tested = nullptr;

    hipMallocAsync(&d_bitfield, sizeof(unsigned long long), stream);
    hipMallocAsync(&d_a, params.M * params.K * sizeof(float), stream);
    hipMallocAsync(&d_b, params.N * params.K * sizeof(float), stream);
    hipMallocAsync(&d_c_sm80, params.M * params.N * sizeof(float), stream);
    hipMallocAsync(&d_c_sm90_warmup, params.M * params.N * sizeof(float), stream);
    hipMallocAsync(&d_c_sm90_tested, params.M * params.N * sizeof(float), stream);

    if (!d_bitfield || !d_a || !d_b || !d_c_sm80 || !d_c_sm90_warmup || !d_c_sm90_tested) {
        fprintf(stderr, "Out of GPU memory\n");
        exit(1);
    }

    // Initialize "random" test data
    {
        dim3 grid_a{(params.K + 15u) / 16u, (params.M + 15u) / 16u, 1};
        dim3 grid_b{(params.N + 15u) / 16u, (params.K + 15u) / 16u, 1};
        dim3 block{16, 16, 1};
        device_init_test_data<<<grid_a, block, 0, stream>>>(d_a, params.M, params.K, params.test_data_code_A);
        device_init_test_data<<<grid_b, block, 0, stream>>>(d_b, params.K, params.N, params.test_data_code_B);
    }

    // Initialize SM80 data
    {
        GPU_Tensors t{params.M, params.N, params.K, d_a, d_b, d_c_sm80};
        matmul_sm80(t, stream);
    }

    // XXX
    {
        gpu_gemm(nullptr, params.M, params.N, params.K, d_a, d_b, d_c_sm90_warmup);
    }
    launch_device_compare_tensor(params, d_c_sm80, d_c_sm90_warmup, params.M, params.N, d_bitfield, stream);

    hipFreeAsync(d_a, stream);
    hipFreeAsync(d_b, stream);
    hipFreeAsync(d_c_sm80, stream);
    hipFreeAsync(d_c_sm90_warmup, stream);
    hipFreeAsync(d_c_sm90_tested, stream);
}
